#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for memory access
__global__ void memory_latency_test(int *data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        int value = data[idx];
        // Perform some operations to ensure memory access
        data[idx] = value + 1;
    }
}

int main() {
    int size = 1; // 1K elements
    int *h_data = (int*)malloc(size * sizeof(int));
    int *d_data;
    hipMalloc(&d_data, size * sizeof(int));

    // Initialize host data
    for (int i = 0; i < size; ++i) {
        h_data[i] = i;
    }

    // Copy data from host to device
    hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);

    // Measure memory latency
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    memory_latency_test<<<(size + 255) / 256, 256>>>(d_data, size);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Memory latency: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipFree(d_data);
    free(h_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}