#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime> 
#include <nvml.h>

// CUDA kernel for memory access
__global__ void memory_latency_test(int *data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        int value = data[idx];
        // Perform some operations to ensure memory access
        data[idx] = (value * 2) + 1;
    }
}

int main() {
    int size = 1024 * 1024 * 1024 * 1; // 1M elements
    int *h_data = (int*)malloc(size * sizeof(int));
    int *d_data;
    hipMalloc(&d_data, size * sizeof(int));

    srand(time(0));
    for (int i = 0; i < size; ++i) {
        h_data[i] = rand();
    }

    // Copy data from host to device
    hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);

    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    unsigned long long energy1, energy2;

    // Measure memory latency
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nvmlDeviceGetTotalEnergyConsumption(device, &energy1);
    hipEventRecord(start);
    memory_latency_test<<<(size + 255) / 256, 256>>>(d_data, size);
    hipEventRecord(stop);
    nvmlDeviceGetTotalEnergyConsumption(device, &energy2);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // long long energyUsage = energy2 - energy1;
    std::cout << "Memory latency: " << milliseconds << " ms" << std::endl;
    std::cout << "Energy usage: " << energy2 << " and " << energy1 << std::endl;

    // Cleanup
    hipFree(d_data);
    free(h_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}